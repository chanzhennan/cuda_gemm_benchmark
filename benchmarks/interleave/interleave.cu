#include "hip/hip_runtime.h"
#include "define_extend/define_extend.cuh"

#define SMEM_LDA (128)
#define SMEM_LDB (128)

// sgemm_128x128x8
__global__ __launch_bounds__(256, 2) void gemm_kernel8(int m, int n, int k,
                                                       const float *a,
                                                       const float *b,
                                                       float *c) {
  __shared__ __align__(
      16 * 1024) char smem[24 * 1024];  // 16KB shared memory for buffer
  float *ashare = reinterpret_cast<float *>(smem);
  float *bshare =
      reinterpret_cast<float *>(smem + 16 * 1024);  // 8k shared mem for B
  float sum[8][8] = {0};
  float panelA[8] = {0}, panelB[8] = {0};

  int from_a = (blockIdx.y * 128 + threadIdx.x / 8 * 4) * k + threadIdx.x % 8;
  int from_b = (threadIdx.x / 32) * n + blockIdx.x * 128 + threadIdx.x % 32;

  for (int loop = 0; loop < k; loop += 8) {
    // part1: gmem to smem
    // load gmem to smem for ashare
    int to_a = (threadIdx.x % 8) * SMEM_LDA +
               (threadIdx.x / 8) * 4;  // 连续的地址不能给同一个 thread 用
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      ashare[to_a + i] = a[from_a + i * k];
    }

    // load gmem to smem for bshare
    int to_b = (threadIdx.x / 32) * SMEM_LDB + (threadIdx.x % 32);
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      bshare[to_b + i * 32] =
          b[from_b + i * 32];  // 32 thread 合并访问。 thread i 访问  [i, i+32,
                               // i+64, i+96]
    }

    __syncthreads();
    from_a += 8;
    from_b += 8 * n;

    // part2: calculation
    // 计算 2x2 个 4x4
    int aidx0 = (threadIdx.x / 16) * 4;
    int bidx0 = (threadIdx.x % 16) * 4;
#pragma unroll
    for (int subk = 0; subk < 8; ++subk) {
      float *ptrA = ashare + aidx0 + subk * SMEM_LDA;

#pragma unroll
      for (int i = 0; i < 4; ++i) {
        panelA[i] = ptrA[i];
        panelA[i + 4] = ptrA[i + 64];
      }

      const float *ptrB = bshare + bidx0 + subk * SMEM_LDB;
#pragma unroll
      for (int i = 0; i < 4; ++i) {
        panelB[i] = ptrB[i];
        panelB[i + 4] = ptrB[i + 64];
      }

#pragma unroll
      for (int i = 0; i < 8; ++i) {
#pragma unroll
        for (int j = 0; j < 8; ++j) {
          sum[i][j] += panelA[i] * panelB[j];
        }
      }
    }
    __syncthreads();
  }

  // part3: save to C
  int write_offset = (blockIdx.y * 128 + (threadIdx.x / 16) * 4) * n +
                     blockIdx.x * 128 + (threadIdx.x % 16) * 4;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    for (int j = 0; j < 4; ++j) {
      c[write_offset + i * n + j] = sum[i][j];
      c[write_offset + i * n + j + 64] = sum[i][j + 4];
      c[write_offset + (i + 64) * n + j] = sum[i + 4][j];
      c[write_offset + (i + 64) * n + j + 64] = sum[i + 4][j + 4];
    }
  }
}

#undef SMEM_LDA
#undef SMEM_LDB

template <typename T>
void GEMM8(T *dA, T *dB, T *dC, int m, int n, int k) {
  constexpr int BLOCK = 128;
  dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);
  gemm_kernel8<<<grid, 256>>>(m, n, k, dA, dB, dC);
  hipDeviceSynchronize();
}

template void GEMM8<float>(float *dA, float *dB, float *dC, int m, int n,
                           int k);
// template void GEMM8<TPB, int>(int *dA, int *dB, int *dC, int m, int n, int
// k);
