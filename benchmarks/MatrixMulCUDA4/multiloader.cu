#include "hip/hip_runtime.h"
#include "MatrixMulCUDA4/multiloader.cuh"

// a = mxk, b = kxn
template <int BLOCK, int STRIDE, typename T>
__global__ void gemm_kernel4(int m, int n, int k, T *a, T *b, T *c) {
  // blockIdx control subpanel matrix
  constexpr int STEP = BLOCK * STRIDE;
  const int tx = threadIdx.x * STRIDE;
  const int ty = threadIdx.y * STRIDE;
  const int bx = blockIdx.x * STEP;
  const int by = blockIdx.y * STEP;

  T *begin_a = a + by * k;
  T *begin_b = b + bx;
  T *end_a = begin_a + k;

  float sum[STRIDE][STRIDE] = {0.f};

  // bigger stash for more data load
  __shared__ T ashare[STEP][2 * STEP];
  __shared__ T bshare[2 * STEP][STEP];

  for (T *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
       a_ptr += 2 * STEP, b_ptr += 2 * STEP * n) {
    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        // load twice each thread
        ashare[ty + i][tx + j] = a_ptr[(ty + i) * k + tx + j];
        ashare[ty + i][tx + j + STEP] = a_ptr[(ty + i) * k + tx + j + STEP];

        bshare[ty + i][tx + j] = b_ptr[(ty + i) * n + tx + j];
        bshare[ty + i + STEP][tx + j] = b_ptr[(ty + i + STEP) * n + tx + j];
      }
    }

    __syncthreads();

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        for (int kk = 0; kk < 2 * STEP; ++kk) {
          sum[i][j] += (float)(ashare[ty + i][kk] * bshare[kk][tx + j]);
        }
      }
    }

    __syncthreads();
  }

#pragma unroll
  for (int i = 0; i < STRIDE; ++i) {
    for (int j = 0; j < STRIDE; ++j) {
      c[(by + ty + i) * n + bx + tx + j] = (T)sum[i][j];
    }
  }
}

template <size_t BLOCK, typename T>
void GEMM4(T *dA, T *dB, T *dC, int m, int n, int k) {
  /*  (BLOCK * BLOCK) threads calc ((BLOCK + STRIDE) * (BLOCK + STRIDE)) data
   *
   *  t0 t1 t0 t1 - - - - - -
   *  - - - - - - - - - - - -
   *  - - - - - - - - - - - -
   *  - - - - - - - - - - - -
   *  - - - - - - - - - - - -
   *  - - - - - - - - - - - -
   *  1. load 2 * 2 float each thread
   *  2. load Gmem -> Smem each thread
   *  3. clac 4 FMA each thread
   *  this kerenl STRIDE = 2
   */

  constexpr int STRIDE = 2;  // every thread calc STRIDExSTRIDE result
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK / STRIDE, (n + BLOCK - 1) / BLOCK / STRIDE);
  gemm_kernel4<BLOCK, STRIDE, T><<<grid, block>>>(m, n, k, dA, dB, dC);
  hipDeviceSynchronize();
}

template void GEMM4<BLOCKSIZE, float>(float *dA, float *dB, float *dC, int m,
                                      int n, int k);

template void GEMM4<BLOCKSIZE, __half>(__half *dA, __half *dB, __half *dC,
                                       int m, int n, int k);
