#include "hip/hip_runtime.h"
#include "MatrixMulCUDA4/multiloader.cuh"

// a = mxk, b = kxn
template <int BLOCK, int STRIDE>
__global__ void gemm_kernel4(int m, int n, int k, float *a, float *b,
                             float *c) {
  // blockIdx control subpanel matrix
  constexpr int STEP = BLOCK * STRIDE;
  const int tx = threadIdx.x * STRIDE;
  const int ty = threadIdx.y * STRIDE;
  const int bx = blockIdx.x * STEP;
  const int by = blockIdx.y * STEP;

  float *begin_a = a + by * k;
  float *begin_b = b + bx;
  float *end_a = begin_a + k;

  float sum[STRIDE][STRIDE] = {0.f};

  // bigger stash for more data load
  __shared__ float ashare[STEP][2 * STEP];
  __shared__ float bshare[2 * STEP][STEP];

  for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
       a_ptr += 2 * STEP, b_ptr += 2 * STEP * n) {
    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        // load twice each thread
        ashare[ty + i][tx + j] = a_ptr[(ty + i) * k + tx + j];
        ashare[ty + i][tx + j + STEP] = a_ptr[(ty + i) * k + tx + j + STEP];

        bshare[ty + i][tx + j] = b_ptr[(ty + i) * n + tx + j];
        bshare[ty + i + STEP][tx + j] = b_ptr[(ty + i + STEP) * n + tx + j];
      }
    }

    __syncthreads();

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        for (int kk = 0; kk < 2 * STEP; ++kk) {
          sum[i][j] += ashare[ty + i][kk] * bshare[kk][tx + j];
        }
      }
    }

    __syncthreads();
  }

#pragma unroll
  for (int i = 0; i < STRIDE; ++i) {
    for (int j = 0; j < STRIDE; ++j) {
      c[(by + ty + i) * n + bx + tx + j] = sum[i][j];
    }
  }
}

template <size_t BLOCK, typename T>
void GEMM4(T *dA, T *dB, T *dC, int m, int n, int k) {
  /*  (BLOCK * BLOCK) threads calc ((BLOCK + STRIDE) * (BLOCK + STRIDE)) data
   *
   *  t0 t1 t0 t1 - - - - - -
   *  - - - - - - - - - - - -
   *  - - - - - - - - - - - -
   *  - - - - - - - - - - - -
   *  - - - - - - - - - - - -
   *  - - - - - - - - - - - -
   *  1. load 2 * 2 float each thread
   *  2. load Gmem -> Smem each thread
   *  3. clac 4 FMA each thread
   *  this kerenl STRIDE = 2
   */

  constexpr int STRIDE = 2;  // every thread calc STRIDExSTRIDE result
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK / STRIDE, (n + BLOCK - 1) / BLOCK / STRIDE);
  gemm_kernel4<BLOCK, STRIDE><<<grid, block>>>(m, n, k, dA, dB, dC);
  hipDeviceSynchronize();
}

template void GEMM4<BLOCKSIZE, float>(float *dA, float *dB, float *dC, int m,
                                      int n, int k);
