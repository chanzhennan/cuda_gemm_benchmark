#include "hip/hip_runtime.h"
// Copyright (c) 2023 Zhennanc Ltd. All rights reserved.
#include <algorithm>
#include <array>
#include <cstdarg>
#include <iostream>
#include <memory>
#include <random>
#include <vector>

#include "utils.h"

namespace cudabm {

std::string strFormatImp(const char* msg, va_list args) {
  // we might need a second shot at this, so pre-emptivly make a copy
  va_list args_cp;
  va_copy(args_cp, args);

  // TODO(ericwf): use std::array for first attempt to avoid one memory
  // allocation guess what the size might be
  std::array<char, 256> local_buff;

  // 2015-10-08: vsnprintf is used instead of snd::vsnprintf due to a limitation
  // in the android-ndk
  auto ret = vsnprintf(local_buff.data(), local_buff.size(), msg, args_cp);

  va_end(args_cp);

  // handle empty expansion
  if (ret == 0) return std::string{};
  if (static_cast<std::size_t>(ret) < local_buff.size())
    return std::string(local_buff.data());

  // we did not provide a long enough buffer on our first attempt.
  // add 1 to size to account for null-byte in size cast to prevent overflow
  std::size_t size = static_cast<std::size_t>(ret) + 1;
  auto buff_ptr = std::unique_ptr<char[]>(new char[size]);
  // 2015-10-08: vsnprintf is used instead of snd::vsnprintf due to a limitation
  // in the android-ndk
  vsnprintf(buff_ptr.get(), size, msg, args);
  return std::string(buff_ptr.get());
}

// adapted from benchmark srcs string utils
std::string strFormat(const char* format, ...) {
  va_list args;
  va_start(args, format);
  std::string tmp = strFormatImp(format, args);
  va_end(args);
  return tmp;
}

void genRandom(std::vector<float>& vec) {
  std::mt19937 gen;
  std::uniform_real_distribution<> dist(-10.0, 10.0);
  std::generate_n(vec.begin(), vec.size(), [&] { return dist(gen); });
}

void genRandom(float* vec, unsigned long len) {
  std::mt19937 gen;
  std::uniform_real_distribution<> dist(-10.0, 10.0);
  for (unsigned long i = 0; i < len; i++) {
    vec[i] = dist(gen);
  }
}

void Print(float* vec, size_t len) {
  for (int i = 0; i < len; i++) {
    printf("%f ", vec[i]);
    if (i % 10 == 0) {
      printf("\n");
    }
  }
}

float Sum(float* vec, size_t len) {
  float sum = 0.f;
  for (int i = 0; i < len; i++) {
    sum += vec[i];
  }
  return sum;
}

void Gemm(float* dA, float* dB, float* dC, int m, int n, int k) {
  float alpha = 1.0f;
  float beta = 0.0f;

  hipblasHandle_t blas_handle;
  hipblasCreate(&blas_handle);

  // if (status != HIPBLAS_STATUS_SUCCESS)
  //   std::runtime_error("!!!! CUBLAS initialization error\n");

  // C = A X B
  hipblasSgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dB, n, dA,
              k, &beta, dC, n);
  hipblasDestroy(blas_handle);
  // hipDeviceSynchronize();
}

// Equal
template <typename Type>
bool Equal(const unsigned int n, const Type* x, const Type* y,
           const Type tolerance) {
  bool ok = true;

  for (unsigned int i = 0; i < n; ++i) {
    if (std::abs(x[i] - y[i]) > std::abs(tolerance)) {
      ok = false;
      return ok;
    }
  }

  return ok;
}

template <typename T, typename S>
int cublas_gemm_ex(hipblasHandle_t handle, hipblasOperation_t transA,
                   hipblasOperation_t transB, int m, int n, int k, T* A, T* B,
                   S* C, int lda, int ldb, int ldc, S* alpha, S* beta,
                   int algo) {
  hipDataType AType, BType, CType, ComputeType;
  if (std::is_same<T, float>::value) {
    AType = BType = CType = ComputeType = HIP_R_32F;
  } else if (std::is_same<T, __half>::value) {
    AType = BType = CType = ComputeType = HIP_R_16F;
  } else if (std::is_same<T, int8_t>::value) {
    AType = BType = HIP_R_8I;
    CType = ComputeType = HIP_R_32I;
  } else {
    printf("Not supported data type.");
    return -1;
  }
  hipblasStatus_t status;
  status = hipblasGemmEx(handle, transA, transB, m, n, k, alpha, A, AType, lda,
                        B, BType, ldb, beta, C, CType, ldc, ComputeType,
                        static_cast<hipblasGemmAlgo_t>(algo));
  if (status == HIPBLAS_STATUS_SUCCESS)
    return 1;
  else
    return -1;
}

template bool Equal<float>(const unsigned int n, const float* x, const float* y,
                           const float tolerance);

}  // namespace cudabm
