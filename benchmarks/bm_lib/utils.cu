#include "hip/hip_runtime.h"
// Copyright (c) 2023 Zhennanc Ltd. All rights reserved.

#include "utils.h"

namespace cudabm {

std::string strFormatImp(const char* msg, va_list args) {
  // we might need a second shot at this, so pre-emptivly make a copy
  va_list args_cp;
  va_copy(args_cp, args);

  // TODO(ericwf): use std::array for first attempt to avoid one memory
  // allocation guess what the size might be
  std::array<char, 256> local_buff;

  // 2015-10-08: vsnprintf is used instead of snd::vsnprintf due to a limitation
  // in the android-ndk
  auto ret = vsnprintf(local_buff.data(), local_buff.size(), msg, args_cp);

  va_end(args_cp);

  // handle empty expansion
  if (ret == 0) return std::string{};
  if (static_cast<std::size_t>(ret) < local_buff.size())
    return std::string(local_buff.data());

  // we did not provide a long enough buffer on our first attempt.
  // add 1 to size to account for null-byte in size cast to prevent overflow
  std::size_t size = static_cast<std::size_t>(ret) + 1;
  auto buff_ptr = std::unique_ptr<char[]>(new char[size]);
  // 2015-10-08: vsnprintf is used instead of snd::vsnprintf due to a limitation
  // in the android-ndk
  vsnprintf(buff_ptr.get(), size, msg, args);
  return std::string(buff_ptr.get());
}

// adapted from benchmark srcs string utils
std::string strFormat(const char* format, ...) {
  va_list args;
  va_start(args, format);
  std::string tmp = strFormatImp(format, args);
  va_end(args);
  return tmp;
}

template <typename T>
void genRandom(T* vec, unsigned long len) {
  std::mt19937 gen;
  std::uniform_real_distribution<> dist(-1.0, 1.0);
  for (unsigned long i = 0; i < len; i++) {
    vec[i] = static_cast<T>(dist(gen));
  }
}

void genOnes(float* vec, unsigned long len) {
  for (unsigned long i = 0; i < len; i++) {
    vec[i] = 1.f;
  }
}

void Print(float* vec, size_t len) {
  for (int i = 0; i < len; i++) {
    printf("%f ", vec[i]);
    if (i % 10 == 0) {
      printf("\n");
    }
  }
}

float Sum(float* vec, size_t len) {
  float sum = 0.f;
  for (int i = 0; i < len; i++) {
    sum += vec[i];
  }
  return sum;
}

template <typename T>
void Gemm(T* dA, T* dB, T* dC, int m, int n, int k) {
  hipblasHandle_t blas_handle;
  hipblasCreate(&blas_handle);

  // C = A X B
  if (std::is_same<T, float>::value) {
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                (float*)dB, m, (float*)dA, k, &beta, (float*)dC, m);

  } else if (std::is_same<T, __half>::value) {
    __half alpha = 1.0f;
    __half beta = 0.0f;
    hipblasHgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                (__half*)dB, m, (__half*)dA, k, &beta, (__half*)dC, m);
  }

  hipblasDestroy(blas_handle);
}

// Equal
template <typename T>
bool Equal(const unsigned int n, const T* x, const T* y,
           const float tolerance) {
  bool ok = true;

  float max_diff = 0.f;
  for (int i = 0; i < n; i++) {
    if (std::abs((float)x[i] - (float)y[i]) > max_diff)
      max_diff = std::abs((float)x[i] - (float)y[i]);
  }

  for (unsigned int i = 0; i < n; ++i) {
    if (std::abs((float)x[i] - (float)y[i]) > std::abs(tolerance)) {
      std::cout << "max_diff " << max_diff << std::endl;
      std::cout << "ours :" << (float)x[i] << " cublas :" << (float)y[i]
                << std::endl;
      ok = false;
      return ok;
    }
  }

  return ok;
}

template bool Equal<float>(const unsigned int n, const float* x, const float* y,
                           const float tolerance);
template bool Equal<half>(const unsigned int n, const half* x, const half* y,
                          const float tolerance);

template void genRandom<float>(float* vec, unsigned long len);
template void genRandom<half>(half* vec, unsigned long len);

template void Gemm<float>(float* dA, float* dB, float* dC, int m, int n, int k);
template void Gemm<__half>(__half* dA, __half* dB, __half* dC, int m, int n,
                           int k);

}  // namespace cudabm
