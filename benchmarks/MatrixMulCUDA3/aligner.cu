#include "hip/hip_runtime.h"
#include "MatrixMulCUDA3/aligner.cuh"

// a = mxk, b = kxn
template <int BLOCK, int STRIDE, typename T>
__global__ void gemm_kernel3(int m, int n, int k, T *a, T *b, T *c) {
  // blockIdx control subpanel matrix
  constexpr int STEP = BLOCK * STRIDE;
  const int tx = threadIdx.x * STRIDE;
  const int ty = threadIdx.y * STRIDE;
  const int bx = blockIdx.x * STEP;
  const int by = blockIdx.y * STEP;

  T *begin_a = a + by * k;
  T *begin_b = b + bx;
  T *end_a = begin_a + k;

  float sum[STRIDE][STRIDE] = {0.f};
  for (T *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
       a_ptr += STEP, b_ptr += STEP * n) {
    //  align shared memory in 16KB
    __shared__ __align__(16 * 1024) T ashare[STEP][STEP];
    __shared__ __align__(16 * 1024) T bshare[STEP][STEP];

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        ashare[ty + i][tx + j] = a_ptr[(ty + i) * k + tx + j];
        bshare[ty + i][tx + j] = b_ptr[(ty + i) * n + tx + j];
      }
    }
    __syncthreads();

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        for (int kk = 0; kk < STEP; ++kk) {
          sum[i][j] += (float)(ashare[ty + i][kk] * bshare[kk][tx + j]);
        }
      }
    }

    __syncthreads();
  }

#pragma unroll
  for (int i = 0; i < STRIDE; ++i) {
    for (int j = 0; j < STRIDE; ++j) {
      c[(by + ty + i) * n + bx + tx + j] = (T)sum[i][j];
    }
  }
}

template <size_t BLOCK, typename T>
void GEMM3(T *dA, T *dB, T *dC, int m, int n, int k) {
  /*  (BLOCK * BLOCK) threads calc ((BLOCK + STRIDE) * (BLOCK + STRIDE)) data
   *
   *  ashared addr
   *  addr(0KB)   addr(16KB)  addr(32KB)  addr(48KB)
   *  t0 t1 t2 t3 t0 t1 t2 t3 t0 t1 t2 t3 t0 t1 t2 t3 t4 t5 ..
   *  - - - - - - - - - - - - - - - - - - - - - - - - - -
   *  - - - - - - - - - - - - - - - - - - - - - - - - - -
   *  - - - - - - - - - - - - - - - - - - - - - - - - - -
   *  - - - - - - - - - - - - - - - - - - - - - - - - - -
   *  - - - - - - - - - - - - - - - - - - - - - - - - - -
   *  1. align shared memory in 16KB
   *  2. clac 4 float(16 byte) each thread
   *  3. load Gmem -> Smem each thread
   *  4. clac 4 FMA each thread
   *  this kerenl STRIDE = 4
   */

  constexpr int STRIDE = 4;  // every thread calc STRIDExSTRIDE result
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK / STRIDE, (n + BLOCK - 1) / BLOCK / STRIDE);
  gemm_kernel3<BLOCK, STRIDE, T><<<grid, block>>>(m, n, k, dA, dB, dC);
  hipDeviceSynchronize();
}

template void GEMM3<BLOCKSIZE, float>(float *dA, float *dB, float *dC, int m,
                                      int n, int k);
template void GEMM3<BLOCKSIZE, __half>(__half *dA, __half *dB, __half *dC,
                                       int m, int n, int k);
