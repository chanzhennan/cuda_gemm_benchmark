#include "hip/hip_runtime.h"
#include "fix_block_size/fix_block_size.cuh"

// a = mxk, b = kxn
template <int BLOCK, int STRIDE>
__global__ void gemm_kernel4(int m, int n, int k, float *a, float *b,
                             float *c) {
  // blockIdx control subpanel matrix
  constexpr int STEP = BLOCK * STRIDE;
  const int tx = threadIdx.x * STRIDE;
  const int ty = threadIdx.y * STRIDE;
  const int bx = blockIdx.x * STEP;
  const int by = blockIdx.y * STEP;

  float *begin_a = a + by * k;
  float *begin_b = b + bx;
  float *end_a = begin_a + k;

  float sum[STRIDE][STRIDE] = {0.f};
  for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
       a_ptr += STEP, b_ptr += STEP * n) {
    __shared__ __align__(16 * 1024) float ashare[STEP][STEP];
    __shared__ __align__(16 * 1024) float bshare[STEP][STEP];

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        ashare[ty + i][tx + j] = a_ptr[(ty + i) * k + tx + j];
        bshare[ty + i][tx + j] = b_ptr[(ty + i) * n + tx + j];
      }
    }
    __syncthreads();

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        for (int kk = 0; kk < STEP; ++kk) {
          sum[i][j] += ashare[ty + i][kk] * bshare[kk][tx + j];
        }
      }
    }

    __syncthreads();
  }

#pragma unroll
  for (int i = 0; i < STRIDE; ++i) {
    for (int j = 0; j < STRIDE; ++j) {
      c[(by + ty + i) * n + bx + tx + j] = sum[i][j];
    }
  }
}

template <size_t BLOCK, typename T>
void GEMM4(T *dA, T *dB, T *dC, int m, int n, int k) {
  constexpr int STRIDE = 4;  // every thread calc STRIDExSTRIDE result
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK / STRIDE, (n + BLOCK - 1) / BLOCK / STRIDE);
  gemm_kernel4<BLOCK, STRIDE><<<grid, block>>>(m, n, k, dA, dB, dC);
  hipDeviceSynchronize();
}

template void GEMM4<TPB, float>(float *dA, float *dB, float *dC, int m, int n,
                                int k);
// template void GEMM4<TPB, int>(int *dA, int *dB, int *dC, int m, int n, int
// k);
